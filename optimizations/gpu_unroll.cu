#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16


__global__ void conv_forward_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_grid = ((W_out - 1) / TILE_WIDTH) + 1;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int height = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y; // # block in row
    int width = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x; // # block in col

    if (height < H_out && width < W_out) {
        float out = 0.0f;
        for (int c = 0; c < C; c++) {

            if (K == 1) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }

            }else if (K == 2) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }
                if (height * S + 0 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 1) * mask_4d(by, c, 0, 1);
                    }
                if (height * S + 1 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 0) * mask_4d(by, c, 1, 0);
                    }
                if (height * S + 1 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 1) * mask_4d(by, c, 1, 1);
                    }

            }else if (K == 3) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }
                if (height * S + 0 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 1) * mask_4d(by, c, 0, 1);
                    }
                if (height * S + 0 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 2) * mask_4d(by, c, 0, 2);
                    }
                if (height * S + 1 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 0) * mask_4d(by, c, 1, 0);
                    }
                if (height * S + 1 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 1) * mask_4d(by, c, 1, 1);
                    }
                if (height * S + 1 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 2) * mask_4d(by, c, 1, 2);
                    }
                if (height * S + 2 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 0) * mask_4d(by, c, 2, 0);
                    }
                if (height * S + 2 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 1) * mask_4d(by, c, 2, 1);
                    }
                if (height * S + 2 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 2) * mask_4d(by, c, 2, 2);
                    }

            }else if (K == 4) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }
                if (height * S + 0 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 1) * mask_4d(by, c, 0, 1);
                    }
                if (height * S + 0 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 2) * mask_4d(by, c, 0, 2);
                    }
                if (height * S + 0 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 3) * mask_4d(by, c, 0, 3);
                    }
                if (height * S + 1 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 0) * mask_4d(by, c, 1, 0);
                    }
                if (height * S + 1 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 1) * mask_4d(by, c, 1, 1);
                    }
                if (height * S + 1 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 2) * mask_4d(by, c, 1, 2);
                    }
                if (height * S + 1 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 3) * mask_4d(by, c, 1, 3);
                    }
                if (height * S + 2 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 0) * mask_4d(by, c, 2, 0);
                    }
                if (height * S + 2 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 1) * mask_4d(by, c, 2, 1);
                    }
                if (height * S + 2 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 2) * mask_4d(by, c, 2, 2);
                    }
                if (height * S + 2 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 3) * mask_4d(by, c, 2, 3);
                    }
                if (height * S + 3 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 0) * mask_4d(by, c, 3, 0);
                    }
                if (height * S + 3 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 1) * mask_4d(by, c, 3, 1);
                    }
                if (height * S + 3 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 2) * mask_4d(by, c, 3, 2);
                    }
                if (height * S + 3 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 3) * mask_4d(by, c, 3, 3);
                    }

            }else if (K == 5) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }
                if (height * S + 0 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 1) * mask_4d(by, c, 0, 1);
                    }
                if (height * S + 0 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 2) * mask_4d(by, c, 0, 2);
                    }
                if (height * S + 0 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 3) * mask_4d(by, c, 0, 3);
                    }
                if (height * S + 0 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 4) * mask_4d(by, c, 0, 4);
                    }
                if (height * S + 1 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 0) * mask_4d(by, c, 1, 0);
                    }
                if (height * S + 1 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 1) * mask_4d(by, c, 1, 1);
                    }
                if (height * S + 1 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 2) * mask_4d(by, c, 1, 2);
                    }
                if (height * S + 1 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 3) * mask_4d(by, c, 1, 3);
                    }
                if (height * S + 1 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 4) * mask_4d(by, c, 1, 4);
                    }
                if (height * S + 2 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 0) * mask_4d(by, c, 2, 0);
                    }
                if (height * S + 2 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 1) * mask_4d(by, c, 2, 1);
                    }
                if (height * S + 2 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 2) * mask_4d(by, c, 2, 2);
                    }
                if (height * S + 2 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 3) * mask_4d(by, c, 2, 3);
                    }
                if (height * S + 2 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 4) * mask_4d(by, c, 2, 4);
                    }
                if (height * S + 3 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 0) * mask_4d(by, c, 3, 0);
                    }
                if (height * S + 3 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 1) * mask_4d(by, c, 3, 1);
                    }
                if (height * S + 3 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 2) * mask_4d(by, c, 3, 2);
                    }
                if (height * S + 3 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 3) * mask_4d(by, c, 3, 3);
                    }
                if (height * S + 3 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 4) * mask_4d(by, c, 3, 4);
                    }
                if (height * S + 4 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 0) * mask_4d(by, c, 4, 0);
                    }
                if (height * S + 4 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 1) * mask_4d(by, c, 4, 1);
                    }
                if (height * S + 4 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 2) * mask_4d(by, c, 4, 2);
                    }
                if (height * S + 4 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 3) * mask_4d(by, c, 4, 3);
                    }
                if (height * S + 4 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 4) * mask_4d(by, c, 4, 4);
                    }

            }else if (K == 6) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }
                if (height * S + 0 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 1) * mask_4d(by, c, 0, 1);
                    }
                if (height * S + 0 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 2) * mask_4d(by, c, 0, 2);
                    }
                if (height * S + 0 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 3) * mask_4d(by, c, 0, 3);
                    }
                if (height * S + 0 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 4) * mask_4d(by, c, 0, 4);
                    }
                if (height * S + 0 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 5) * mask_4d(by, c, 0, 5);
                    }
                if (height * S + 1 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 0) * mask_4d(by, c, 1, 0);
                    }
                if (height * S + 1 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 1) * mask_4d(by, c, 1, 1);
                    }
                if (height * S + 1 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 2) * mask_4d(by, c, 1, 2);
                    }
                if (height * S + 1 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 3) * mask_4d(by, c, 1, 3);
                    }
                if (height * S + 1 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 4) * mask_4d(by, c, 1, 4);
                    }
                if (height * S + 1 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 5) * mask_4d(by, c, 1, 5);
                    }
                if (height * S + 2 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 0) * mask_4d(by, c, 2, 0);
                    }
                if (height * S + 2 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 1) * mask_4d(by, c, 2, 1);
                    }
                if (height * S + 2 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 2) * mask_4d(by, c, 2, 2);
                    }
                if (height * S + 2 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 3) * mask_4d(by, c, 2, 3);
                    }
                if (height * S + 2 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 4) * mask_4d(by, c, 2, 4);
                    }
                if (height * S + 2 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 5) * mask_4d(by, c, 2, 5);
                    }
                if (height * S + 3 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 0) * mask_4d(by, c, 3, 0);
                    }
                if (height * S + 3 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 1) * mask_4d(by, c, 3, 1);
                    }
                if (height * S + 3 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 2) * mask_4d(by, c, 3, 2);
                    }
                if (height * S + 3 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 3) * mask_4d(by, c, 3, 3);
                    }
                if (height * S + 3 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 4) * mask_4d(by, c, 3, 4);
                    }
                if (height * S + 3 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 5) * mask_4d(by, c, 3, 5);
                    }
                if (height * S + 4 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 0) * mask_4d(by, c, 4, 0);
                    }
                if (height * S + 4 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 1) * mask_4d(by, c, 4, 1);
                    }
                if (height * S + 4 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 2) * mask_4d(by, c, 4, 2);
                    }
                if (height * S + 4 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 3) * mask_4d(by, c, 4, 3);
                    }
                if (height * S + 4 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 4) * mask_4d(by, c, 4, 4);
                    }
                if (height * S + 4 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 5) * mask_4d(by, c, 4, 5);
                    }
                if (height * S + 5 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 0) * mask_4d(by, c, 5, 0);
                    }
                if (height * S + 5 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 1) * mask_4d(by, c, 5, 1);
                    }
                if (height * S + 5 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 2) * mask_4d(by, c, 5, 2);
                    }
                if (height * S + 5 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 3) * mask_4d(by, c, 5, 3);
                    }
                if (height * S + 5 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 4) * mask_4d(by, c, 5, 4);
                    }
                if (height * S + 5 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 5) * mask_4d(by, c, 5, 5);
                    }

            }else if (K == 7) {
                if (height * S + 0 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 0) * mask_4d(by, c, 0, 0);
                    }
                if (height * S + 0 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 1) * mask_4d(by, c, 0, 1);
                    }
                if (height * S + 0 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 2) * mask_4d(by, c, 0, 2);
                    }
                if (height * S + 0 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 3) * mask_4d(by, c, 0, 3);
                    }
                if (height * S + 0 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 4) * mask_4d(by, c, 0, 4);
                    }
                if (height * S + 0 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 5) * mask_4d(by, c, 0, 5);
                    }
                if (height * S + 0 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 0, width * S + 6) * mask_4d(by, c, 0, 6);
                    }
                if (height * S + 1 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 0) * mask_4d(by, c, 1, 0);
                    }
                if (height * S + 1 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 1) * mask_4d(by, c, 1, 1);
                    }
                if (height * S + 1 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 2) * mask_4d(by, c, 1, 2);
                    }
                if (height * S + 1 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 3) * mask_4d(by, c, 1, 3);
                    }
                if (height * S + 1 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 4) * mask_4d(by, c, 1, 4);
                    }
                if (height * S + 1 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 5) * mask_4d(by, c, 1, 5);
                    }
                if (height * S + 1 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 1, width * S + 6) * mask_4d(by, c, 1, 6);
                    }
                if (height * S + 2 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 0) * mask_4d(by, c, 2, 0);
                    }
                if (height * S + 2 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 1) * mask_4d(by, c, 2, 1);
                    }
                if (height * S + 2 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 2) * mask_4d(by, c, 2, 2);
                    }
                if (height * S + 2 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 3) * mask_4d(by, c, 2, 3);
                    }
                if (height * S + 2 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 4) * mask_4d(by, c, 2, 4);
                    }
                if (height * S + 2 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 5) * mask_4d(by, c, 2, 5);
                    }
                if (height * S + 2 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 2, width * S + 6) * mask_4d(by, c, 2, 6);
                    }
                if (height * S + 3 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 0) * mask_4d(by, c, 3, 0);
                    }
                if (height * S + 3 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 1) * mask_4d(by, c, 3, 1);
                    }
                if (height * S + 3 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 2) * mask_4d(by, c, 3, 2);
                    }
                if (height * S + 3 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 3) * mask_4d(by, c, 3, 3);
                    }
                if (height * S + 3 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 4) * mask_4d(by, c, 3, 4);
                    }
                if (height * S + 3 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 5) * mask_4d(by, c, 3, 5);
                    }
                if (height * S + 3 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 3, width * S + 6) * mask_4d(by, c, 3, 6);
                    }
                if (height * S + 4 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 0) * mask_4d(by, c, 4, 0);
                    }
                if (height * S + 4 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 1) * mask_4d(by, c, 4, 1);
                    }
                if (height * S + 4 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 2) * mask_4d(by, c, 4, 2);
                    }
                if (height * S + 4 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 3) * mask_4d(by, c, 4, 3);
                    }
                if (height * S + 4 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 4) * mask_4d(by, c, 4, 4);
                    }
                if (height * S + 4 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 5) * mask_4d(by, c, 4, 5);
                    }
                if (height * S + 4 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 4, width * S + 6) * mask_4d(by, c, 4, 6);
                    }
                if (height * S + 5 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 0) * mask_4d(by, c, 5, 0);
                    }
                if (height * S + 5 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 1) * mask_4d(by, c, 5, 1);
                    }
                if (height * S + 5 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 2) * mask_4d(by, c, 5, 2);
                    }
                if (height * S + 5 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 3) * mask_4d(by, c, 5, 3);
                    }
                if (height * S + 5 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 4) * mask_4d(by, c, 5, 4);
                    }
                if (height * S + 5 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 5) * mask_4d(by, c, 5, 5);
                    }
                if (height * S + 5 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 5, width * S + 6) * mask_4d(by, c, 5, 6);
                    }
                if (height * S + 6 < H && width * S + 0 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 0) * mask_4d(by, c, 6, 0);
                    }
                if (height * S + 6 < H && width * S + 1 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 1) * mask_4d(by, c, 6, 1);
                    }
                if (height * S + 6 < H && width * S + 2 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 2) * mask_4d(by, c, 6, 2);
                    }
                if (height * S + 6 < H && width * S + 3 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 3) * mask_4d(by, c, 6, 3);
                    }
                if (height * S + 6 < H && width * S + 4 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 4) * mask_4d(by, c, 6, 4);
                    }
                if (height * S + 6 < H && width * S + 5 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 5) * mask_4d(by, c, 6, 5);
                    }
                if (height * S + 6 < H && width * S + 6 < W) {
                    out += in_4d(bx, c, height * S + 6, width * S + 6) * mask_4d(by, c, 6, 6);
                    }

            }else{
                for (int p=0; p<K; p++) {
                    for (int q=0; q<K; q++) {
                        if (height * S + p < H && width * S + q < W) {
                            out += in_4d(bx, c, height * S + p, width * S + q) * mask_4d(by, c, p, q);
                        }
                    }
                }
            }
        }
        out_4d(bx, by, height, width) = out;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float * __restrict__ host_output, const float * __restrict__ host_input, const float * __restrict__ host_mask, float ** __restrict__ device_output_ptr, float ** __restrict__ device_input_ptr, float ** __restrict__ device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    int in_size = B * C * H * W;
    int out_size = B * M * H_out * W_out;
    int mask_size = M * C * K * K;

    hipMalloc((void**) device_input_ptr, in_size* sizeof(float));
    hipMalloc((void**) device_output_ptr, out_size* sizeof(float));
    hipMalloc((void**) device_mask_ptr, mask_size* sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, in_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size*sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float * __restrict__ device_output, const float * __restrict__ device_input, const float * __restrict__ device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    int H_grid = ((H_out - 1)/ TILE_WIDTH) + 1;
    int W_grid = ((W_out - 1)/ TILE_WIDTH) + 1;
    int Z_gird = H_grid * W_grid;

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(B, M, Z_gird);

    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S); 

    
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float * __restrict__ host_output, float * __restrict__ device_output, float * __restrict__ device_input, float * __restrict__ device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int out_size = B * M * H_out * W_out;

    hipMemcpy(host_output, device_output, out_size * sizeof(float), hipMemcpyDeviceToHost);  
    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}